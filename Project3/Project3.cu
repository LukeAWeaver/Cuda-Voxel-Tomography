#include "hip/hip_runtime.h"
#include <iostream>
#include<stdio.h>
#include "ImageWriter.h"
#include<hip/hip_runtime.h>
#include <string>
#include <fstream>
using namespace std;


__global__ void kernel_MAX(int* pixelDepth)
{
	pixelDepth[0] = 222;
}
__global__ void kernel_SUM(unsigned char* voxelData,int pixelDepth)
{
}

void kernelHandler(int nRows, int nCols, int nSheets, string inFile, int projectionType)
{
	int relativeHeight;
	int relativeWidth;
	int h_relativeDepth;
	unsigned char* raw_voxelData = new unsigned char[nRows*nSheets*nCols]();
	ifstream s(inFile, ios::binary);
	s.read(reinterpret_cast<char*>(raw_voxelData), nCols*nRows*nSheets); 
	s.close();
	unsigned char* h_voxelData = new unsigned char[nRows*nSheets*nCols]();
	switch(projectionType)
	{
		case 1: //0 degree roation 0ol n = n
		relativeHeight = nRows;
		relativeWidth = nCols;
		h_relativeDepth = nSheets;
		break;
		case 2: //180 degree rotation horizontally
		relativeHeight = nRows;
		relativeWidth = nCols;
		h_relativeDepth = nSheets;
		for(int s = 0; s<nSheets;s++)
		{
			for(int i = 0; i<nCols; i++)
			{
				for(int j = 0; j<nRows;j++)
				{
					int currentSheet = j*i*s;
					int relativeValue = j+j*i;
					int originalPerspective = (nRows-j-1) + ((nRows-j-1)*(nCols-i-1));
					h_voxelData[relativeValue + currentSheet] = raw_voxelData[(nRows-j-1) + originalPerspective + currentSheet];
				}
			}
		}
		break;
		case 3: //90 degree rotation horizontally clockwise
		relativeHeight = nSheets;
		relativeWidth = nRows;
		h_relativeDepth = nCols;
		for(int s = 0; s<nSheets;s++)
		{
			for(int i = 0; i<nCols; i++)
			{
				for(int j = 0; j<nRows;j++)
				{
					int currentSheet = j*i*s;
					int relativeValue = j+j*i;
					int originalPerspective = (nRows-j-1) + ((nRows-j-1)*(nCols-i-1));
					h_voxelData[relativeValue + currentSheet] = raw_voxelData[(nRows-j-1) + originalPerspective + currentSheet];
				}
			}
		}
		break;
		case 4: //-90 degree rotation horizontally counterclockwise
		relativeHeight = nSheets;
		relativeWidth = nRows;
		h_relativeDepth = nCols;
		break;
		case 5://90 degree rotation upward
		relativeHeight = nCols;
		relativeWidth = nSheets;
		h_relativeDepth = nRows;
		break;
		case 6: // 90 degree rotation download
		relativeHeight = nCols;
		relativeWidth = nSheets;
		h_relativeDepth = nRows;
		break;
	}
	
	unsigned char *d_voxelData;
	
	int a = 5;
	int* temp = new int[1];
	temp[0] = a;
	int* d_relativeDepth;

	cout << "Before kernel: " << temp[0] << endl;

	//size_t size = nRows*nCols*nSheets*sizeof(char);
	//hipMalloc((void**)&d_voxelData,size);
	hipMalloc((void**)&d_relativeDepth,sizeof(int));
	//hipMemcpy(d_voxelData,raw_voxelData,size,hipMemcpyHostToDevice);
	hipMemcpy(d_relativeDepth,temp,sizeof(int),hipMemcpyHostToDevice);
	// Invoke kernel
	kernel_MAX<<<relativeWidth,relativeHeight>>>(d_relativeDepth);
	//kernel_SUM<<<relativeWidth,relativeHeight>>>(d_voxelData,temp);
	hipDeviceSynchronize();
	// Copy result from device memory to host memory
	//hipMemcpy(h_voxelData,d_voxelData,size,hipMemcpyDeviceToHost);
	hipMemcpy(temp,d_relativeDepth,sizeof(int),hipMemcpyDeviceToHost);
	// Free device memory
	cout << "Before kernel: " << temp[0] << endl;
	hipFree(d_relativeDepth);
}

void writeTheFile(string outFile, int xres, int yres, const unsigned char* imageBytes)
{
	unsigned char* row = new unsigned char[3*xres];
	ImageWriter *w = ImageWriter:: create(outFile,xres,yres);
	int next = 0;
	for(int r = 0; r<yres; r++)
	{
		for(int c = 0; c<3*xres; c+=3)
		{
			row[c] = row[c+1] = row[c+2] = imageBytes[next++];
		}
		w->addScanLine(row);
	}
	w->closeImageFile();
	delete w;
	delete[] row;
}

int main(int argc, char* argv[])
{
	int nRows = atoi( argv[1] );
	int nCols = atoi(argv[2]);
	int nSheets = atoi(argv[3]);
	string inFile = argv[4];
	int pt = stoi(argv[5]);
	string outFile = argv[6];
	kernelHandler(nRows,nCols,nSheets,inFile,pt);
	hipError_t err = hipDeviceSynchronize();
	if ( err != hipSuccess )
	{
		printf("%s", hipGetErrorString(err));
	}
	return 0;
}
